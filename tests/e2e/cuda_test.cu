#include <stdio.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(err)); \
            return 1; \
        } \
    } while (0)

// Test parameters
#define DATA_SIZE (1024 * 1024 * 256)  // 256MB of data
#define BLOCK_SIZE 256
#define NUM_ITERATIONS 100

__global__ void vectorAdd(const float *A, const float *B, float *C, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    float *h_A, *h_B, *h_C;    // Host arrays
    float *d_A, *d_B, *d_C;    // Device arrays
    int size = DATA_SIZE;
    size_t bytes = size * sizeof(float);
    hipEvent_t start, stop;
    float milliseconds;

    printf("Anarchy eGPU CUDA Test\n");
    printf("=====================\n");

    // Allocate host memory
    h_A = (float*)malloc(bytes);
    h_B = (float*)malloc(bytes);
    h_C = (float*)malloc(bytes);

    // Initialize host arrays
    for (int i = 0; i < size; i++) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    // Create CUDA events for timing
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // Allocate device memory
    CHECK_CUDA(hipMalloc(&d_A, bytes));
    CHECK_CUDA(hipMalloc(&d_B, bytes));
    CHECK_CUDA(hipMalloc(&d_C, bytes));

    printf("\nMemory Transfer Test\n");
    printf("-------------------\n");

    // Test memory transfer (Host to Device)
    CHECK_CUDA(hipEventRecord(start));
    for (int i = 0; i < NUM_ITERATIONS; i++) {
        CHECK_CUDA(hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice));
    }
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    
    float h2d_bandwidth = (2.0f * bytes * NUM_ITERATIONS) / (milliseconds * 1000000.0f);
    printf("Host to Device Bandwidth: %.2f GB/s\n", h2d_bandwidth);

    // Launch kernel
    int threadsPerBlock = BLOCK_SIZE;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    printf("\nComputation Test\n");
    printf("----------------\n");

    CHECK_CUDA(hipEventRecord(start));
    for (int i = 0; i < NUM_ITERATIONS; i++) {
        vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, size);
    }
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));

    float compute_throughput = (2.0f * size * NUM_ITERATIONS) / (milliseconds * 1000000.0f);
    printf("Computation Throughput: %.2f GFLOPS\n", compute_throughput);

    // Test memory transfer (Device to Host)
    CHECK_CUDA(hipEventRecord(start));
    for (int i = 0; i < NUM_ITERATIONS; i++) {
        CHECK_CUDA(hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost));
    }
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));

    float d2h_bandwidth = (bytes * NUM_ITERATIONS) / (milliseconds * 1000000.0f);
    printf("Device to Host Bandwidth: %.2f GB/s\n", d2h_bandwidth);

    // Verify results
    printf("\nVerification Test\n");
    printf("----------------\n");
    bool error = false;
    for (int i = 0; i < size; i++) {
        if (fabs(h_C[i] - (h_A[i] + h_B[i])) > 1e-5) {
            fprintf(stderr, "Verification failed at index %d\n", i);
            error = true;
            break;
        }
    }
    if (!error) {
        printf("All computations verified successfully!\n");
    }

    // Cleanup
    free(h_A);
    free(h_B);
    free(h_C);
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));

    return error ? 1 : 0;
} 